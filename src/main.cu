#include "hip/hip_runtime.h"
#include <fstream>
#include <iomanip>
#include <iostream>
#include <map>

#include "material/diffuse_light.h"
#include "material/solid_color.h"
#include "material/texture/checker_texture.h"
#include "material/texture/image_texture.h"
#include "material/texture/noise_texture.h"
#include "object/aa_rectangle.h"
#include "object/box.h"
#include "object/bvh.h"
#include "object/camera.h"
#include "object/constant_medium.h"
#include "object/hittable_list.h"
#include "object/moving_sphere.h"
#include "object/rotate.h"
#include "object/sphere.h"
#include "object/translate.h"
#include "utility/color.h"
#include "utility/rtweekend.h"

#pragma clang diagnostic push
#pragma ide diagnostic ignored "misc-no-recursion"
Color RayColor(const Ray& r, const Color& background, const Hittable& world,
               int depth) {
  HitRecord hit_record;

  // If we've exceeded the ray bounce limit, no more light is gathered.
  if (depth <= 0) {
    return {0, 0, 0};
  }
  // If the ray hits nothing, return the background color.
  if (!world.Hit(r, 0.001, infinity, &hit_record)) {
    return background;
  }

  Ray scattered;
  Color attenuation;
  Color emitted =
      hit_record.material->Emitted(hit_record.u, hit_record.v, hit_record.p);

  if (!hit_record.material->Scatter(r, hit_record, &attenuation, &scattered)) {
    return emitted;
  }

  return emitted +
         attenuation * RayColor(scattered, background, world, depth - 1);
}
#pragma clang diagnostic pop

HittableList RandomScene(shared_ptr<Camera> camera, bool has_time = true,
                         bool has_checker_texture = true) {
  HittableList boxes;
  HittableList world;

  double time0 = 0.0f;
  double time1 = 1.0f;
  if (!has_time) {
    time0 = 0.0f;
    time1 = 0.0f;
  }

  if (has_checker_texture) {
    auto checker =
        make_shared<CheckTexture>(make_shared<SolidColor>(0.2f, 0.3f, 0.1f),
                                  make_shared<SolidColor>(0.9f, 0.9f, 0.9f));
    boxes.Add(make_shared<Sphere>(Point3(0, -1000, 0), 1000,
                                  make_shared<Lambertian>(checker)));
  } else {
    auto ground_material = make_shared<Lambertian>(Color(0.5, 0.5, 0.5));
    boxes.Add(make_shared<Sphere>(Point3(0, -1000, 0), 1000, ground_material));
  }

  for (int a = -11; a < 11; a++) {
    for (int b = -11; b < 11; b++) {
      auto choose_mat = RandomDouble();
      Point3 center(a + 0.9 * RandomDouble(), 0.2, b + 0.9 * RandomDouble());

      if ((center - Point3(4, 0.2, 0)).Length() > 0.9) {
        shared_ptr<Material> sphere_material;

        if (choose_mat < 0.8) {
          // diffuse
          auto albedo = Color::Random() * Color::Random();
          sphere_material = make_shared<Lambertian>(albedo);
          auto center2 = center + Vec3(0, RandomDouble(0, 0.5), 0);
          boxes.Add(make_shared<MovingSphere>(center, center2, time0, time1,
                                              0.2, sphere_material));
        } else if (choose_mat < 0.95) {
          // metal
          auto albedo = Color::Random(0.5, 1);
          auto fuzz = RandomDouble(0, 0.5);
          sphere_material = make_shared<Metal>(albedo, fuzz);
          boxes.Add(make_shared<Sphere>(center, 0.2, sphere_material));
        } else {
          // glass
          sphere_material = make_shared<Dielectric>(1.5);
          boxes.Add(make_shared<Sphere>(center, 0.2, sphere_material));
        }
      }
    }
  }

  auto material1 = make_shared<Dielectric>(1.5);
  boxes.Add(make_shared<Sphere>(Point3(0, 1, 0), 1.0, material1));

  auto material2 = make_shared<Lambertian>(Color(0.4, 0.2, 0.1));
  boxes.Add(make_shared<Sphere>(Point3(-4, 1, 0), 1.0, material2));

  auto material3 = make_shared<Metal>(Color(0.7, 0.6, 0.5), 0.0);
  boxes.Add(make_shared<Sphere>(Point3(4, 1, 0), 1.0, material3));

  world.Add(make_shared<BvhNode>(boxes, 0, 1));

  world.camera_ = camera;
  return world;
}

HittableList TwoSpheres(shared_ptr<Camera> camera) {
  HittableList objects;

  auto checker =
      make_shared<CheckTexture>(make_shared<SolidColor>(0.2f, 0.3f, 0.1f),
                                make_shared<SolidColor>(0.9f, 0.9f, 0.9f));
  objects.Add(make_shared<Sphere>(Point3(0, -10, 0), 10,
                                  make_shared<Lambertian>(checker)));
  objects.Add(make_shared<Sphere>(Point3(0, 10, 0), 10,
                                  make_shared<Lambertian>(checker)));

  objects.camera_ = camera;

  return objects;
}

HittableList TwoPerlinSpheres(shared_ptr<Camera> camera) {
  HittableList objects;

  auto per_text = make_shared<NoiseTexture>(4);
  objects.Add(make_shared<Sphere>(Point3(0, -1000, 0), 1000,
                                  make_shared<Lambertian>(per_text)));
  objects.Add(make_shared<Sphere>(Point3(0, 2, 0), 2,
                                  make_shared<Lambertian>(per_text)));

  objects.camera_ = camera;
  return objects;
}

HittableList Earth(shared_ptr<Camera> camera) {
  auto earth_texture = make_shared<ImageTexture>("resources/earth-map.jpg");
  auto earth_surface = make_shared<Lambertian>(earth_texture);
  auto globe = make_shared<Sphere>(Point3(0, 0, 0), 2, earth_surface);
  auto world = HittableList(globe);

  world.camera_ = camera;

  return world;
}

HittableList SampleLight(std::shared_ptr<Camera>& camera) {
  HittableList objects;
  auto per_text = make_shared<NoiseTexture>(4);
  objects.Add(make_shared<Sphere>(Point3(0, -1000, 0), 1000,
                                  make_shared<Lambertian>(per_text)));
  objects.Add(make_shared<Sphere>(Point3(0, 2, 0), 2,
                                  make_shared<Lambertian>(per_text)));

  auto diff_light = make_shared<DiffuseLight>(make_shared<SolidColor>(4, 4, 4));
  objects.Add(make_shared<Sphere>(Point3(0, 7, 0), 2, diff_light));
  objects.Add(make_shared<XyRectangle>(3, 5, 1, 3, -2, diff_light));

  Point3 look_from(26.0, 3.0, 6.0);
  Point3 look_at(0, 2, 0);
  auto aperture = 0.01;

  objects.camera_ =
      make_shared<Camera>(look_from, look_at, camera->v_up_, camera->v_fov_,
                          camera->aspect_ratio_, aperture, camera->focus_dist_);

  return objects;
}

HittableList CornellBox(std::shared_ptr<Camera>& camera) {
  HittableList objects;

  auto red = std::make_shared<Lambertian>(Color(0.65, 0.05, 0.05));
  auto white = std::make_shared<Lambertian>(Color(0.73, 0.73, 0.73));
  auto green = std::make_shared<Lambertian>(Color(.12, .45, .15));
  auto light = std::make_shared<DiffuseLight>(Color(15, 15, 15));

  objects.Add(std::make_shared<YzRectangle>(0, 555, 0, 555, 555, green));
  objects.Add(std::make_shared<YzRectangle>(0, 555, 0, 555, 0, red));
  objects.Add(std::make_shared<XzRectangle>(213, 343, 227, 332, 554, light));
  objects.Add(std::make_shared<XzRectangle>(0, 555, 0, 555, 0, white));
  objects.Add(std::make_shared<XzRectangle>(0, 555, 0, 555, 555, white));
  objects.Add(std::make_shared<XyRectangle>(0, 555, 0, 555, 555, white));

  std::shared_ptr<Hittable> box1 =
      std::make_shared<Box>(Point3(0, 0, 0), Point3(165, 330, 165), white);
  box1 = make_shared<RotateY>(box1, 15);
  box1 = make_shared<Translate>(box1, Vec3(265, 0, 295));
  objects.Add(box1);

  std::shared_ptr<Hittable> box2 =
      std::make_shared<Box>(Point3(0, 0, 0), Point3(165, 165, 165), white);
  box2 = make_shared<RotateY>(box2, -18);
  box2 = make_shared<Translate>(box2, Vec3(130, 0, 65));
  objects.Add(box2);

  objects.camera_ = std::make_shared<Camera>(
      Point3(278, 278, -800), Point3(278, 278, 0), camera->v_up_, 40, 1.0,
      camera->aperture_, camera->focus_dist_, Color(0, 0, 0), 0, 0);

  return objects;
}

HittableList CornellSmoke(std::shared_ptr<Camera>& camera) {
  HittableList objects;

  auto red = std::make_shared<Lambertian>(Color(0.65, 0.05, 0.05));
  auto white = std::make_shared<Lambertian>(Color(0.73, 0.73, 0.73));
  auto green = std::make_shared<Lambertian>(Color(.12, .45, .15));
  auto light = std::make_shared<DiffuseLight>(Color(7, 7, 7));

  objects.Add(std::make_shared<YzRectangle>(0, 555, 0, 555, 555, green));
  objects.Add(std::make_shared<YzRectangle>(0, 555, 0, 555, 0, red));
  objects.Add(std::make_shared<XzRectangle>(113, 443, 127, 432, 554, light));
  objects.Add(std::make_shared<XzRectangle>(0, 555, 0, 555, 0, white));
  objects.Add(std::make_shared<XzRectangle>(0, 555, 0, 555, 555, white));
  objects.Add(std::make_shared<XyRectangle>(0, 555, 0, 555, 555, white));

  std::shared_ptr<Hittable> box1 =
      std::make_shared<Box>(Point3(0, 0, 0), Point3(165, 330, 165), white);
  box1 = make_shared<RotateY>(box1, 15);
  box1 = make_shared<Translate>(box1, Vec3(265, 0, 295));

  std::shared_ptr<Hittable> box2 =
      std::make_shared<Box>(Point3(0, 0, 0), Point3(165, 165, 165), white);
  box2 = make_shared<RotateY>(box2, -18);
  box2 = make_shared<Translate>(box2, Vec3(130, 0, 65));

  objects.Add(make_shared<ConstantMedium>(box1, 0.01, Color(0, 0, 0)));
  objects.Add(make_shared<ConstantMedium>(box2, 0.01, Color(1, 1, 1)));

  objects.camera_ = std::make_shared<Camera>(
      Point3(278, 278, -800), Point3(278, 278, 0), camera->v_up_, 40, 1.0,
      camera->aperture_, camera->focus_dist_, Color(0, 0, 0), 0, 0);

  return objects;
}

HittableList TheNextWeek(std::shared_ptr<Camera>& camera) {
  HittableList boxes1;
  auto ground = make_shared<Lambertian>(Color(0.48, 0.83, 0.53));

  const int boxes_per_side = 20;
  for (int i = 0; i < boxes_per_side; i++) {
    for (int j = 0; j < boxes_per_side; j++) {
      auto w = 100.0;
      auto x0 = -1000.0 + i * w;
      auto z0 = -1000.0 + j * w;
      auto y0 = 0.0;
      auto x1 = x0 + w;
      auto y1 = RandomDouble(1, 101);
      auto z1 = z0 + w;

      boxes1.Add(
          make_shared<Box>(Point3(x0, y0, z0), Point3(x1, y1, z1), ground));
    }
  }

  HittableList objects;

  objects.Add(make_shared<BvhNode>(boxes1, 0, 1));

  auto light = make_shared<DiffuseLight>(Color(7, 7, 7));
  objects.Add(make_shared<XzRectangle>(123, 423, 147, 412, 554, light));

  auto center1 = Point3(400, 400, 200);
  auto center2 = center1 + Vec3(30, 0, 0);
  auto moving_sphere_material = make_shared<Lambertian>(Color(0.7, 0.3, 0.1));
  objects.Add(make_shared<MovingSphere>(center1, center2, 0, 1, 50,
                                        moving_sphere_material));

  objects.Add(make_shared<Sphere>(Point3(260, 150, 45), 50,
                                  make_shared<Dielectric>(1.5)));
  objects.Add(make_shared<Sphere>(
      Point3(0, 150, 145), 50, make_shared<Metal>(Color(0.8, 0.8, 0.9), 1.0)));

  auto boundary = make_shared<Sphere>(Point3(360, 150, 145), 70,
                                      make_shared<Dielectric>(1.5));
  objects.Add(boundary);
  objects.Add(make_shared<ConstantMedium>(boundary, 0.2, Color(0.2, 0.4, 0.9)));
  boundary =
      make_shared<Sphere>(Point3(0, 0, 0), 5000, make_shared<Dielectric>(1.5));
  objects.Add(make_shared<ConstantMedium>(boundary, .0001, Color(1, 1, 1)));

  auto e_mat = make_shared<Lambertian>(
      make_shared<ImageTexture>("resources/earth-map.jpg"));
  objects.Add(make_shared<Sphere>(Point3(400, 200, 400), 100, e_mat));
  auto per_text = make_shared<NoiseTexture>(0.1);
  objects.Add(make_shared<Sphere>(Point3(220, 280, 300), 80,
                                  make_shared<Lambertian>(per_text)));

  HittableList boxes2;
  auto white = make_shared<Lambertian>(Color(.73, .73, .73));
  int ns = 1000;
  for (int j = 0; j < ns; j++) {
    boxes2.Add(make_shared<Sphere>(Point3::Random(0, 165), 10, white));
  }

  objects.Add(make_shared<Translate>(
      make_shared<RotateY>(make_shared<BvhNode>(boxes2, 0.0, 1.0), 15),
      Vec3(-100, 270, 395)));

  auto new_camera = std::make_shared<Camera>(
      Point3(478, 278, -600), Point3(278, 278, 0), camera->v_up_, 40, 1.0,
      camera->aperture_, camera->focus_dist_, Color(0, 0, 0), 0, 1);

  camera = new_camera;

  return objects;
}

void Render(unsigned int i, unsigned int j, Vec3* fb, int image_width,
            int image_height, const HittableList& world, int max_depth,
            int samples_per_pixel) {
  if ((i >= image_width) || (j >= image_height)) {
    return;
  }
  Color pixel_color(0, 0, 0);
  auto camera = world.camera_;

  for (int s = 0; s < samples_per_pixel; ++s) {
    auto u = (i + RandomDouble()) / (image_width - 1);
    auto v = (j + RandomDouble()) / (image_height - 1);

    Ray ray = camera->GetRay(u, v);
    auto background = camera->background_;

    pixel_color += RayColor(ray, background, world, max_depth);
  }

  unsigned int pixel_index = j * image_width + i;
  fb[pixel_index] = pixel_color;
}

int main(int argc, char** argv) {
  // Camera
  auto aspect_ratio = 16.0 / 9.0;
  Point3 look_from(13.0, 2.0, 3.0);
  Point3 look_at(0, 0, 0);
  Vec3 v_up(0, 1, 0);
  auto dist_to_focus = 10.0;
  auto aperture = 0.1;
  auto v_fov = 20.0;

  auto camera = std::make_shared<Camera>(look_from, look_at, v_up, v_fov,
                                         aspect_ratio, aperture, dist_to_focus,
                                         Color(0.70, 0.80, 1.00), 0.0f, 1.0f);

  // World
  auto world_map = std::map<std::string, HittableList>{
      {"Random", RandomScene(camera, false, false)},
      {"WithTime", RandomScene(camera, true, false)},
      {"CheckerTexture", RandomScene(camera, true, true)},
      {"TwoSpheres", TwoSpheres(camera)},
      {"TwoPerlinSpheres", TwoPerlinSpheres(camera)},
      {"Earth", Earth(camera)},
      {"SampleLight", SampleLight(camera)},
      {"CornellBox", CornellBox(camera)},
      {"CornellSmoke", CornellSmoke(camera)},
      {"TheNextWeek", TheNextWeek(camera)},
  };

  // Image
  int image_width = 1600;
  int samples_per_pixel = 500;
  const int max_depth = 50;
  std::string scene_name = "Random";

  // Read Environment Variables
  if (const char* env_p = std::getenv("SPP")) {
    samples_per_pixel = std::stoi(env_p);
  }
  if (const char* env_p = std::getenv("SCENE")) {
    scene_name = env_p;
  }
  if (const char* env_p = std::getenv("IMAGE_WIDTH")) {
    image_width = std::stoi(env_p);
  }

  if (world_map.find(scene_name) == world_map.end()) {
    std::cerr << "Scene " << scene_name << " not found" << std::endl;
    return 1;
  }
  std::cerr << "Rendering Scene:  " << scene_name << std::endl;
  auto world = world_map[scene_name];

  camera = world.camera_;
  aspect_ratio = camera->aspect_ratio_;
  int image_height = static_cast<int>(image_width / aspect_ratio);

  // Output
  std::ofstream ofs(scene_name + ".ppm");

  // Render
  ofs << "P3\n" << image_width << ' ' << image_height << "\n255\n";

  auto* fb = new Color[image_width * image_height];

  clock_t start, stop;
  start = clock();

  for (int j = image_height - 1; j >= 0; --j) {
    std::cerr << "\rScanline's remaining:" << j << ' ' << std::flush;
    for (int i = 0; i < image_width; ++i) {
      Render(i, j, fb, image_width, image_height, world, max_depth,
             samples_per_pixel);
    }
  }

  stop = clock();
  double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
  std::cerr << std::endl << "Took " << timer_seconds << " seconds.\n";

  for (int j = image_height - 1; j >= 0; --j) {
    for (int i = 0; i < image_width; ++i) {
      auto pixel_index = j * image_width + i;
      write_color(ofs, fb[pixel_index], samples_per_pixel);
    }
  }

  std::cerr << "\nDone.\n";
}
